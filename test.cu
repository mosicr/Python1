#define N (2048*2048)
#define THREADS_PER_BLOCK 512


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

// GPU kernel function to add two vectors
__global__ void add_gpu( int *a, int *b, int *c, int n){
  int index = threadIdx.x + blockIdx.x * blockDim.x;
    if (index < n)
        c[index] = a[index] + b[index];
	}

// CPU function to add two vectors
void add_cpu (int *a, int *b, int *c, int n) {
  for (int i=0; i < n; i++)
      c[i] = a[i] + b[i];
      }

// CPU function to generate a vector of random integers
void random_ints (int *a, int n) {
  for (int i = 0; i < n; i++)
    a[i] = rand() % 10000; // random number between 0 and 9999
    }

// CPU function to compare two vectors
int compare_ints( int *a, int *b, int n ){
  int pass = 0;
    for (int i = 0; i < N; i++){
        if (a[i] != b[i]) {
	      printf("Value mismatch at location %d, values %d and %d\n",i, a[i], b[i]);
	            pass = 1;
		        }
			  }

if (pass == 0) printf ("Test passed\n"); else printf ("Test Failed\n");
			      return pass;
			      }


int main( void ) {

    int *a, *b, *c; // host copies of a, b, c
    int *dev_a, *dev_b, *dev_c; // device copies of a, b, c
    int size = N * sizeof( int ); // we need space for N integers
//    printf("N = %d\n", N);
  // Allocate GPU/device copies of dev_a, dev_b, dev_c
    hipMalloc( (void**)&dev_a, size );
    hipMalloc( (void**)&dev_b, size );
    hipMalloc( (void**)&dev_c, size );

  // Allocate CPU/host copies of a, b, c
    a = (int*)malloc( size );
    b = (int*)malloc( size );
    c = (int*)malloc( size );

  // Fill input vectors with random integer numbers
    random_ints( a, N );
    random_ints( b, N );
/*    printf("a = %d\n", a[2048]);
      printf("a = %d\n", a[2]);
      printf("b = %d\n", b[1]);
      printf("b = %d\n", b[2]);
*/
// copy inputs to device
    hipMemcpy( dev_a, a, size, hipMemcpyHostToDevice );
    hipMemcpy( dev_b, b, size, hipMemcpyHostToDevice );

  // launch add_gpu() kernel with blocks and threads
    add_gpu<<< N/THREADS_PER_BLOCK, THREADS_PER_BLOCK >>>( dev_a, dev_b, dev_c, N );

  // copy device result back to host copy of c
    hipMemcpy( c, dev_c, size, hipMemcpyDeviceToHost );
    printf("c_gpu %d\n", *c);
  //Check the results with CPU implementation
    int *c_h; c_h = (int*)malloc( size );
      add_cpu (a, b, c_h, N);
    printf("c_host %d\n", *c_h);
      
        compare_ints(c, c_h, N);

  // Clean CPU memory allocations
    free( a ); free( b ); free( c ); free (c_h);

  // Clean GPU memory allocations
    hipFree( dev_a );
      hipFree( dev_b );
        hipFree( dev_c );

  return 0;
  }